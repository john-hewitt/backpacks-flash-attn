// Adapted from https://github.com/NVIDIA/apex/blob/master/csrc/fused_dense_cuda.cu
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11000
// includes cublaslt
#include <hipblaslt.h>
#endif

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    at::Half* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// BF16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    at::BFloat16* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_16BF,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600

int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void* gelu_in,
    const void* bias) {
  bool save_gelu_in = gelu_in != nullptr;
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (save_gelu_in) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));
  }

  if (bias != nullptr) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS : HIPBLASLT_EPILOGUE_GELU_BIAS;
  } else {
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          // &heuristicResult.algo,
                          // TD [2022-04-29] Somehow algo 0 and 2 are a lot slower than other algos
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void* gelu_in,
    const void* bias) {
  bool save_gelu_in = gelu_in != nullptr;
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (save_gelu_in) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));
  }

  if (bias != nullptr) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS : HIPBLASLT_EPILOGUE_GELU_BIAS;
  } else {
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          // &heuristicResult.algo,
                          // TD [2022-04-29] Somehow algo 0 and 2 are a lot slower than other algos
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (bgrad != nullptr) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (bgrad != nullptr) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

#endif

template <typename T>
int linear_bias_wgrad_cuda(T *input, T *d_output, int in_features, int batch_size, int out_features, T *d_weight, T *d_bias, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero      = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    in_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    input,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_weight,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    static_cast<const void*>(d_bias));
#endif

    if (status != 0){
        status = gemm_bias(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          in_features,
          out_features,
          batch_size,
          &alpha,
          input,
          in_features,
          d_output,
          out_features,
          &beta_zero,
          d_weight,
          in_features);
    }

    return status;
}

template <typename T>
int linear_gelu_forward_cuda(T *input, T *weight, T *bias, int in_features, int batch_size, int out_features, int heuristic, T *output, T *gelu_in, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_gelu_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    input,
    in_features,
    &beta_zero, /* host pointer */
    output,
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    heuristic,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(bias));
    return status;
#else
    return 1;
#endif
}

template <typename T>
int bias_gelu_linear_dgrad_bgrad_cuda(T *weight, T *d_output, T *gelu_in, int in_features, int batch_size, int out_features, int heuristic, T *d_input, T *d_bias, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero      = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_dgelu_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    in_features,
    batch_size,
    out_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_input,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    heuristic,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(d_bias));
#endif
    return status;

}

template int linear_bias_wgrad_cuda<at::Half>(at::Half *input, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, void *lt_workspace) ;
template int linear_bias_wgrad_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, int in_features, int batch_size, int out_features, at::BFloat16 *d_weight, at::BFloat16 *d_bias, void *lt_workspace) ;

template int linear_gelu_forward_cuda<at::Half>(at::Half *input, at::Half *weight, at::Half *bias, int in_features, int batch_size, int out_features, int heuristic, at::Half *output, at::Half *gelu_in, void *lt_workspace) ;
template int linear_gelu_forward_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *weight, at::BFloat16 *bias, int in_features, int batch_size, int out_features, int heuristic, at::BFloat16 *output, at::BFloat16 *gelu_in, void *lt_workspace) ;

template int bias_gelu_linear_dgrad_bgrad_cuda<at::Half>(at::Half *weight, at::Half *d_output, at::Half *gelu_in, int in_features, int batch_size, int out_features, int heuristic, at::Half *d_input, at::Half *d_bias, void *lt_workspace);
template int bias_gelu_linear_dgrad_bgrad_cuda<at::BFloat16>(at::BFloat16 *weight, at::BFloat16 *d_output, at::BFloat16 *gelu_in, int in_features, int batch_size, int out_features, int heuristic, at::BFloat16 *d_input, at::BFloat16 *d_bias, void *lt_workspace);